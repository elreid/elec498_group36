/*
Development of the CheckSum Feature and Timestamps.

CHECK-DONE/SUM




TIMESTAMPS
- few methods of doing. NOTE: TIMING THE THREADS.
    1. we could use a cuda event
    2. use something like clock()
    3. clock in kernel code?


  ------------------------------  


  cudaEvent_t start, stop;
    cudaEventCreate(&start);
    cudaEventCreate(&stop);
    cudaEventRecord(start);

    <some kernel here>    

    cudaEventRecord(stop);
    cudaEventSynchronize(stop);
    float gpuTime = 0.0f;
    cudaEventElapsedTime(&gpuTime, start, stop);
    cout << "Time to complete: " << gpuTime << " milliseconds" << endl;

    ----------------------
    3. clock in kernel code:
    t1 = myCPUTimer();
    t2 = myCPUTimer();





*/



#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <iostream>
#include <time.h>
#include <sys/time.h>

#define USECPSEC 1000000ULL

//definition of number of threads per block. 


//TIMING KERNEL EXECUTION WITH CPU TIMERS:
unsigned long long myCPUTimer(unsigned long long start=0){

  timeval tv;
  gettimeofday(&tv, 0);
  return ((tv.tv_sec*USECPSEC)+tv.tv_usec)-start;
} //returns time in microseconds 

//https://stackoverflow.com/questions/7876624/timing-cuda-operations
//https://stackoverflow.com/questions/69136940/timing-kernel-execution-with-cpu-timers

//TO-DO:
//1. breakdown of how many elements for the checksum.
//2. similarly, need an array for keeping track of times. can store together. 

