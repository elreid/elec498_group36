#define NUMNODES 5

void conversion(Node *head, int* IOVector, int length){
    Node *current = head;

    int i = 0;

    while(current != NULL && i < length*3){

        IOVector[i] = current-> size;
        IOVector[i+1] = current->partition;
        IOVector[i+2] = current->buffer;
        current = current->next;
        i+= 3;
    }
}

int *h_io = (int *)malloc(NUMNODES*3*sizeof(int));
int* d_io;

hipMalloc((void**)&d_io, (NUMNODES*3*sizeof(int)));

conversion(*head, *h_io, NUMNODES);

cudamemcpy(d_io,h_io, NUMNODES*3*sizeof(int), hipMemcpyHostToDevice);
