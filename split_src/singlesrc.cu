#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <unistd.h>
#include <time.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#include "hip/hip_runtime_api.h"



#define N 16
#define USECPSEC 1000000ULL
#define NUMPARTITIONS 4
#define NUMNODES 5000

__global__ void matrixAddition(int *A, int *B, int *C, int size)
{
	// if (threadIdx.x == 0)
	// {
	// 	printf("[MAT_ADD]: Ping from block %d, thread %d\n", blockIdx.x, threadIdx.x);
	// }

	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	if (row < size && col < size)
	{
		int temp = row * size + col;
		C[temp] = A[temp] + B[temp];
	} // close if
}

