#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <unistd.h>
#include <time.h>
#include <sys/time.h>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"


// #include "cuPrintf.cu"

#define TPB 16 // num threads in a block
#define D 256  // num of elements in a row/column
#define N 16
#define USECPSEC 1000000ULL
#define NUMPARTITIONS 4
#define NUMNODES 5


// GLOBAL CHECKSUM VARIABLE
int  CHECKSUM[NUMNODES] = {0};

// GLOBAL FLAG VARIABLE 
int flag = 0;

/***
 * @brief From "forvanya.txt"
 */
void printArray(int *array, int length)
{
	for (int i = 0; i < length; i++)
	{
		if (i % 3 == 0)
			printf("\n");
		printf("%08X ", array[i]);
	}
	printf("\n");
}

void hostAddition(int *A, int *B, int *C, int size)
{
	for (int i = 0; i < size; i++)
	{
		for (int j = 0; j < size; j++)
		{
			C[i * size + j] = A[i * size + j] + B[i * size + j];
		}
	}
} // close hostaddition

__global__ void matrixAddition(int *A, int *B, int *C, int size)
{

	if (threadIdx.x == 0)
	{
		printf("[MAT_ADD]: Ping from block %d, thread %d\n", blockIdx.x, threadIdx.x);
	}

	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	if (row < size && col < size)
	{
		int temp = row * size + col;
		C[temp] = A[temp] + B[temp];
	} // close if
}

/**
 * @brief Master kernel for checksum flaggin
 *
 *
 */
__global__ void master_kernel(int *d_arr, int *check_sum, int num_nodes)
{
	check_sum[num_nodes - 1] = 1;
}
/**
 * @brief Test kernel
 *  - for printing functionality
 *
 */
__global__ void print_kernel()
{
	int i = 0; 
	i = i + 1;
	for(int j = 0; j < 100; j++){
		j = j+i;
	}
}

void myStreamCallback(hipStream_t event, hipError_t status, void *data)
{

	int *check_sum = (int *) data ;
	check_sum[0] = 1;
	printf("Callback function called\n");
	flag = 1;

}
/**
 * @brief Launching the master kernel with the params. from cpu.c
 */
extern "C" void launch_master(int *d_arr, int *check_sum, int num_nodes)
{

	dim3 threadsPerBlock(TPB, TPB);
	dim3 numberOfBlocks(ceil(D / threadsPerBlock.x), ceil(D / threadsPerBlock.y));

	/***
	 * @brief Creating streams for each node
	 * Undefined number of streams
	 */

	hipStream_t streams[num_nodes];

	//***
	// @brief Creating streams for each node
	for (int i = 0; i < num_nodes; i++)
	{
		hipError_t response;

		response = hipStreamCreate(&streams[i]);
		if(response != hipSuccess){
			printf("[ERROR]: Stream creation failed for stream %d\n", i);
			printf("\t- CUDA error: %s\n", hipGetErrorString(response));
		}
		
		response = hipStreamAddCallback(streams[i], myStreamCallback, check_sum, 0);
		if(response != hipSuccess){
			printf("[ERROR]: Attaching callback function failed for stream %d\n", i);
			printf("\t- CUDA error: %s\n", hipGetErrorString(response));
		}

		printf("\n");

	}

	//***
	// @brief Wiring up the kernels to their specific streams
	for (int i = 0; i < num_nodes; i++)
	{
		print_kernel<<<1, 1, 0, streams[i]>>>();
	}
	hipDeviceSynchronize();

	//***
	// @brief Destroying the streams
	for (int i = 0; i < num_nodes; i++)
	{
		hipStreamDestroy(streams[i]);
	}

	printf("Flag: %d\n", flag);

	printf("Finished launching master function\n");
}

extern "C" void launch_matrix_multiply()
{
	/**
	 * @brief Doing the matrix multiplication
	 *
	 */
	time_t t;
	hipEvent_t start, stop, start1, stop1;

	hipEventCreate(&start);
	hipEventCreate(&start1);

	hipEventCreate(&stop);
	hipEventCreate(&stop1);

	float gpu_time = 0.0f, gpu_time1 = 0.0;

	size_t size = D * D * sizeof(int);

	// create pointers for host related stuff, allocate the memory required
	int *h_A = (int *)malloc(size);
	int *h_B = (int *)malloc(size);
	int *h_C = (int *)malloc(size);
	int *h_C1 = (int *)malloc(size);

	// create pointers for device related stuff, allocate the memory required
	int *d_A, *d_B, *d_C;
	hipMalloc((void **)&d_A, size);
	hipMalloc((void **)&d_B, size);
	hipMalloc((void **)&d_C, size);

	// seed that THICC BOI
	srand((unsigned)time(&t));

	// send in values into the host 2 input matrices
	for (int i = 0; i < D; i++)
	{
		for (int j = 0; j < D; j++)
		{
			int rand1 = rand() % 10;
			int rand2 = rand() % 10;
			*(h_A + i * D + j) = rand1;
			*(h_B + i * D + j) = rand2;
		}
	}

	// run it back baby, host addition start, stop and method call
	// general function timing // banya stuff
	clock_t start_test, end_test;
	double cpu_time_used;
	start_test = clock();
	// banya stuff
	hipEventRecord(start, 0);
	hostAddition(h_A, h_B, h_C, D);
	hipEventRecord(stop, 0);

	// print out the results
	hipEventElapsedTime(&gpu_time, start, stop);
	printf("host addition time:\t\t%0.2f\n", gpu_time);
	// banya stuff
	end_test = clock();
	cpu_time_used = ((double)(end_test - start_test));
	printf("(banya) h add time :\t\t%0.2f\n", cpu_time_used);
	// banya stuff

	// copy contents of host input matrices to the device
	hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

	// setup threads per block and number of blocks.
	// should change D to just be strictly 16 later based on documentation ??...
	dim3 threadsPerBlock(TPB, TPB);
	dim3 numberOfBlocks(ceil(D / threadsPerBlock.x), ceil(D / threadsPerBlock.y));

	// addition by individual threads:
	start_test = clock();
	//
	hipEventRecord(start1, 0);
	matrixAddition<<<numberOfBlocks, threadsPerBlock>>>(d_A, d_B, d_C, D);

	hipEventRecord(stop1, 0);
	hipEventSynchronize(stop1);
	hipMemcpy(h_C1, d_C, size, hipMemcpyDeviceToHost);
	hipEventElapsedTime(&gpu_time1, start1, stop1);
	printf("normal matrix addition:\t\t%0.2f\n", gpu_time1);
	//
	end_test = clock();
	cpu_time_used = ((double)(end_test - start_test));
	printf("(banya) norm mat add :\t\t%0.2f\n", cpu_time_used);
	//
}


int main(int argc, char **argv)
{
	// launch_matrix_multiply();
	size_t size_list_arr = (NUMNODES*3)*sizeof(int);

	int *d_list_arr;
    hipMalloc( (void**) &d_list_arr , size_list_arr );

	launch_master(d_list_arr, CHECKSUM, NUMNODES);

	return 0;
}