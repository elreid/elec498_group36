#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <unistd.h>
#include <time.h>
#include <sys/time.h>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include <hip/hip_runtime.h>
#include ""
// #include "cuPrintf.cu"

#define TPB 16 // num threads in a block
#define D 1024 // num of elements in a row/column
#define N 16
#define USECPSEC 1000000ULL
#define NUMPARTITIONS 4

// GLOBAL CHECKSUM VARIABLE
// int  CHECKSUM[NUMNODES] = {0};
struct workload
{
	int *data_arr;
	int *check_sum;
	int id;
	int numnodes;
};

// GLOBAL FLAG VARIABLE
int flag = 0;

// Global Time Variables
time_t t;
clock_t start_test, end_test;

/***
 * @brief From "forvanya.txt"
 */
void printArray(int *array, int length)
{
	for (int i = 0; i < length; i++)
	{
		if (i % 3 == 0)
			printf("\n");
		printf("%08X ", array[i]);
	}
	printf("\n");
}

void hostAddition(int *A, int *B, int *C, int size)
{
	for (int i = 0; i < size; i++)
	{
		for (int j = 0; j < size; j++)
		{
			C[i * size + j] = A[i * size + j] + B[i * size + j];
		}
	}
} // close hostaddition

__global__ void matrixAddition(int *A, int *B, int *C, int size)
{
	// if (threadIdx.x == 0)
	// {
	// 	printf("[MAT_ADD]: Ping from block %d, thread %d\n", blockIdx.x, threadIdx.x);
	// }

	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	if (row < size && col < size)
	{
		int temp = row * size + col;
		C[temp] = A[temp] + B[temp];
	} // close if
}

/**
 * @brief Master kernel for checksum flaggin
 *
 *
 */
__global__ void master_kernel(int *d_arr, int *check_sum, int num_nodes)
{
	check_sum[num_nodes - 1] = 1;
}
/**
 * @brief Test kernel
 *  - for printing functionality
 *
 */
__global__ void print_kernel()
{
	int i = 0;
	i = i + 1;
	for (int j = 0; j < 100; j++)
	{
		j = j + i;
	}
}

void CUDART_CB myStreamCallback(hipStream_t event, hipError_t status, void *data)
{
	printf("============================================\n");
	
	struct workload *workload = (struct workload *)data;

	printf("Workload ID: [%d],  Event: [%08X]\n", workload->id, event);
	if (status != hipSuccess)
		printf("ERR: %s\n", hipGetErrorString(status));
	
	
	workload->check_sum[workload->id] = 1;

	workload->data_arr[workload->id * 3] = 0xACCED000 | workload->id;

	printf("Checksum: ");
	for (int i = 0; i < workload->numnodes; i++)
	{
		printf("%d ", workload->check_sum[i]);
	}
	printf(", Time Finished: %0.2f\n", (double)(clock() - start_test));
	printf("============================================\n");
	printf("\n\n");
}
/**
 * @brief Launching the master kernel with the params. from cpu.c
 */
extern "C" void launch_master(int *data_arr, int *check_sum, int num_nodes)
{
	srand((unsigned)time(&t));
	start_test = clock();

	dim3 threadsPerBlock(TPB, TPB);
	dim3 numberOfBlocks(ceil(D / threadsPerBlock.x), ceil(D / threadsPerBlock.y));

	/***
	 * Checking if the d_arr is passed over correctly
	 */
	printf("[LAUNCH_MASTER]: Printing the data_arr\n");
	printArray(data_arr, 3 * num_nodes);
	printf("\n");

	/***
	 * @brief Creating streams for each node
	 * Undefined number of streams
	 */
	hipStream_t streams[num_nodes];

	//***
	// @brief Creating streams for each node
	for (int i = 0; i < num_nodes; i++)
	{
		hipError_t response;

		response = hipStreamCreate(&streams[i]);
		if (response != hipSuccess)
		{
			printf("[ERROR]: Stream creation failed for stream %d\n", i);
			printf("\t- CUDA error: %s\n", hipGetErrorString(response));
		}
		else
		{
			printf("Stream %d created as [%08X]\n", i, streams[i]);
		}

		/**
		 * @brief
		 *  Creating the workload and attaching the callback function to the stream
		 */
		workload *workload = (struct workload *)malloc(sizeof(struct workload));

		workload->data_arr = data_arr;
		workload->check_sum = check_sum;
		workload->numnodes = num_nodes;
		workload->id = i;

		response = hipStreamAddCallback(streams[i], myStreamCallback, workload, 0);
		if (response != hipSuccess)
		{
			printf("[ERROR]: Attaching callback function failed for stream %d\n", i);
			printf("\t- CUDA error: %s\n", hipGetErrorString(response));
		}
		else
		{
			printf("Callback function attached to stream %d, Object: [%08X]\n", i, streams[i]);
		}
		/**
		 * End of cb_
		 *
		 */
	}
	printf("\n\n");

	//***
	// @brief Wiring up the kernels to their specific streams
	for (int i = 0; i < num_nodes; i++)
	{

		size_t size = D * D * sizeof(int);

		// create pointers for host related stuff, allocate the memory required
		int *h_A = (int *)malloc(size);
		int *h_B = (int *)malloc(size);
		int *h_C = (int *)malloc(size);
		int *h_C1 = (int *)malloc(size);

		// create pointers for device related stuff, allocate the memory required
		int *d_A, *d_B, *d_C;
		hipMalloc((void **)&d_A, size);
		hipMalloc((void **)&d_B, size);
		hipMalloc((void **)&d_C, size);

		// send in values into the host 2 input matrices, randomness in the arrays
		for (int x = 0; x < D; x++)
		{
			for (int y = 0; y < D; y++)
			{
				int rand1 = rand() % 10;
				int rand2 = rand() % 10;
				*(h_A + x * D + y) = rand1;
				*(h_B + x * D + y) = rand2;
			}
		}
		/**
		 * @brief Construct a new cuda Memcpy Async object
		 *
		 */
		// copy contents of host input matrices to the device
		hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, streams[i]);
		hipMemcpyAsync(d_B, h_B, size, hipMemcpyHostToDevice, streams[i]);
		/**
		 * @brief kernel inst.
		 *
		 * INSTANTIATE THE KERNEL
		 *
		 */
		matrixAddition<<<1, 1, streams[i]>>>(d_A, d_B, d_C, D);
		// print_kernel<<<1, 1, 0, streams[i]>>>();
	}
	printf("\n\n");

	hipDeviceSynchronize();

	//***
	// @brief Destroying the streams
	// for (int i = 0; i < num_nodes; i++)
	// {
	// 	hipStreamDestroy(streams[i]);
	// }

	// printf("Checksum: ");
	// for (int i = 0; i < num_nodes; i++){
	// 	printf("%d ", check_sum[i]);
	// }
	// printf("\n");

	printf("Finished launching master function\n");
}

void launch_bogus()
{
	dim3 threadsPerBlock(TPB, TPB);
	dim3 numberOfBlocks(ceil(D / threadsPerBlock.x), ceil(D / threadsPerBlock.y));

	hipStream_t stream1, stream2, stream3;
	hipError_t response;
	response = hipStreamCreate(&stream1);
	printf("CUDA error: %s, %d\n", hipGetErrorString(response), response);
	response = hipStreamCreate(&stream2);
	printf("CUDA error: %s, %d\n", hipGetErrorString(response), response);
	response = hipStreamCreate(&stream3);
	printf("CUDA error: %s, %d\n", hipGetErrorString(response), response);

	for (int i = 0; i < 100; i++)
	{
		print_kernel<<<numberOfBlocks, threadsPerBlock, 0, stream1>>>();
		print_kernel<<<numberOfBlocks, threadsPerBlock, 0, stream2>>>();
		print_kernel<<<numberOfBlocks, threadsPerBlock, 0, stream3>>>();
		hipDeviceSynchronize();
	}
}

extern "C" void launch_matrix_multiply()
{
	/**
	 * @brief Doing the matrix multiplication
	 *
	 */
	time_t t;
	hipEvent_t start, stop, start1, stop1;

	hipEventCreate(&start);
	hipEventCreate(&start1);

	hipEventCreate(&stop);
	hipEventCreate(&stop1);

	float gpu_time = 0.0f, gpu_time1 = 0.0;

	size_t size = D * D * sizeof(int);

	// create pointers for host related stuff, allocate the memory required
	int *h_A = (int *)malloc(size);
	int *h_B = (int *)malloc(size);
	int *h_C = (int *)malloc(size);
	int *h_C1 = (int *)malloc(size);

	// create pointers for device related stuff, allocate the memory required
	int *d_A, *d_B, *d_C;
	hipMalloc((void **)&d_A, size);
	hipMalloc((void **)&d_B, size);
	hipMalloc((void **)&d_C, size);

	// seed that THICC BOI
	srand((unsigned)time(&t));

	// send in values into the host 2 input matrices
	for (int i = 0; i < D; i++)
	{
		for (int j = 0; j < D; j++)
		{
			int rand1 = rand() % 10;
			int rand2 = rand() % 10;
			*(h_A + i * D + j) = rand1;
			*(h_B + i * D + j) = rand2;
		}
	}

	// run it back baby, host addition start, stop and method call
	// general function timing // banya stuff
	clock_t start_test, end_test;
	double cpu_time_used;
	start_test = clock();
	// banya stuff
	hipEventRecord(start, 0);
	hostAddition(h_A, h_B, h_C, D);
	hipEventRecord(stop, 0);

	// print out the results
	hipEventElapsedTime(&gpu_time, start, stop);
	printf("host addition time:\t\t%0.2f\n", gpu_time);
	// banya stuff
	end_test = clock();
	cpu_time_used = ((double)(end_test - start_test));
	printf("(banya) h add time :\t\t%0.2f\n", cpu_time_used);
	// banya stuff

	// copy contents of host input matrices to the device
	hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

	// setup threads per block and number of blocks.
	// should change D to just be strictly 16 later based on documentation ??...
	dim3 threadsPerBlock(TPB, TPB);
	dim3 numberOfBlocks(ceil(D / threadsPerBlock.x), ceil(D / threadsPerBlock.y));

	// addition by individual threads:
	start_test = clock();
	//
	hipEventRecord(start1, 0);
	matrixAddition<<<numberOfBlocks, threadsPerBlock>>>(d_A, d_B, d_C, D);

	hipEventRecord(stop1, 0);
	hipEventSynchronize(stop1);
	hipMemcpy(h_C1, d_C, size, hipMemcpyDeviceToHost);
	hipEventElapsedTime(&gpu_time1, start1, stop1);
	printf("normal matrix addition:\t\t%0.2f\n", gpu_time1);
	//
	end_test = clock();
	cpu_time_used = ((double)(end_test - start_test));
	printf("(banya) norm mat add :\t\t%0.2f\n", cpu_time_used);
	//
}

// int main(int argc, char **argv)
// {
// 	// launch_matrix_multiply();
// 	size_t size_list_arr = (NUMNODES*3)*sizeof(int);

// 	int *d_list_arr;
//     hipMalloc( (void**) &d_list_arr , size_list_arr );

// 	// launch_master(d_list_arr, CHECKSUM, NUMNODES);

// 	launch_bogus();

// 	return 0;
// }