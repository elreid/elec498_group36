#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <unistd.h>
#include <time.h>
#include <sys/time.h>
#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include ""

#define TPB 16	//num threads in a block
#define D 256	//num of elements in a row/column

/***
 * @brief From "forvanya.txt"
*/
void printArray(int *array, int length)
{
    for (int i = 0; i < length; i++)
    {
        if (i % 3 == 0)
            printf("\n");
        printf("%08X ", array[i]);
    }
    printf("\n");
}

void hostAddition(int *A, int *B, int *C, int size) 
{ 
	for (int i = 0; i < size; i++) {
		for (int j = 0; j < size; j++) {
			C[i*size + j] = A[i*size + j] + B[i*size + j];
		}
	}
}//close hostaddition

__global__ void matrixAddition(int *A, int *B, int *C, int size) {
	int row = blockIdx.y*blockDim.y + threadIdx.y;
	int col = blockIdx.x*blockDim.x + threadIdx.x;

	if (row < size && col < size) {
		int temp = row * size + col;
		C[temp] = A[temp] + B[temp];
	}//close if
}

/**
 * @brief Master kernel for checksum flaggin
 * 
 * 
 */
__global__ void master_kernel(int * d_arr, int * check_sum, int num_nodes)
{
	check_sum[num_nodes-1] = 1;
}


/**
 * @brief Launching the master kernel with the params. from cpu.c
 */
extern "C" void launch_master(int * d_arr, int * check_sum, int num_nodes)
{
	// printf("d_arr: %X\n", d_arr);
	// for(int i = 0; i < num_nodes*3; i++){
	// 	printf("d_arr[%d]: %d\n", i, d_arr[i]);
	// }
	// for(int i = 0; i < num_nodes; i++){
	// 	printf("check_sum[%d]: %d\n", i, check_sum[i]);
	// 	if (i==3){
	// 		check_sum[i] = 1;
	// 	}
	// }
	// dim3 threadsPerBlock(TPB, TPB);
	// dim3 numberOfBlocks(ceil(D / threadsPerBlock.x), ceil(D / threadsPerBlock.y));

	// master_kernel <<<numberOfBlocks, threadsPerBlock>>>(d_arr, check_sum, num_nodes);
	// printArray(check_sum, num_nodes);
	// check_sum[num_nodes-1] = 1;

	cudaStrem_t stream_arr[num_nodes];

   	for(int i=0;i<num_nodes;i++){

		hipStream_t stream
		stream_arr[i] = stream;
   		hipStreamCreate(&stream)
    	matrix_add<<<blocks,threads,0,stream>>>(d_a,d_b);
    	
  	}
	
	hipDeviceSynchronize();


}

extern "C" void launch_matrix_multiply()
{
    /**
     * @brief Doing the matrix multiplication
     * 
     */
    time_t t;
    hipEvent_t start, stop, start1, stop1;

    hipEventCreate(&start);
	hipEventCreate(&start1);

    hipEventCreate(&stop);
	hipEventCreate(&stop1);

    float gpu_time = 0.0f, gpu_time1 = 0.0;

    size_t size = D*D*sizeof(int);

    //create pointers for host related stuff, allocate the memory required
	int *h_A 	= (int*)malloc(size);
	int *h_B 	= (int*)malloc(size);
	int *h_C 	= (int*)malloc(size);
	int *h_C1 	= (int*)malloc(size);

	//create pointers for device related stuff, allocate the memory required
	int *d_A, *d_B, *d_C;
	hipMalloc((void**)&d_A, size);
	hipMalloc((void**)&d_B, size);
	hipMalloc((void**)&d_C, size);

	//seed that THICC BOI
	srand((unsigned)time(&t));

    //send in values into the host 2 input matrices
	for (int i = 0; i < D; i++) {
		for (int j = 0; j < D; j++) {
			int rand1 = rand() % 10;
			int rand2 = rand() % 10;
			*(h_A + i * D + j) = rand1;
			*(h_B + i * D + j) = rand2;
		}
	}

    //run it back baby, host addition start, stop and method call
	//general function timing // banya stuff 
	clock_t start_test, end_test;
	double cpu_time_used;
	start_test = clock();
	// banya stuff
	hipEventRecord(start, 0);
	hostAddition(h_A, h_B, h_C, D);
	hipEventRecord(stop, 0);

	//print out the results
	hipEventElapsedTime(&gpu_time, start, stop);
	printf("host addition time:\t\t%0.2f\n", gpu_time);
	// banya stuff
	end_test = clock();
    cpu_time_used = ((double) (end_test - start_test));
	printf("(banya) h add time :\t\t%0.2f\n", cpu_time_used);
	// banya stuff

	//copy contents of host input matrices to the device
	hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    //setup threads per block and number of blocks.
	//should change D to just be strictly 16 later based on documentation ??...
	dim3 threadsPerBlock(TPB, TPB);
	dim3 numberOfBlocks(ceil(D / threadsPerBlock.x), ceil(D / threadsPerBlock.y));

	//addition by individual threads:
	start_test = clock();
	//
	hipEventRecord(start1, 0);
	matrixAddition <<<numberOfBlocks, threadsPerBlock>>>(d_A, d_B, d_C, D);
	hipEventRecord(stop1, 0);
	hipEventSynchronize(stop1);
	hipMemcpy(h_C1, d_C, size, hipMemcpyDeviceToHost);
	hipEventElapsedTime(&gpu_time1, start1, stop1);
	printf("normal matrix addition:\t\t%0.2f\n", gpu_time1);
	//
	end_test = clock();
    cpu_time_used = ((double) (end_test - start_test));
	printf("(banya) norm mat add :\t\t%0.2f\n", cpu_time_used);
	//

}