#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <unistd.h>
#include <time.h>
#include <sys/time.h>
#include "hip/hip_runtime.h"



#define TPB 16	//num threads in a block
#define D 256

/***
 * @brief From "forvanya.txt"
*/
void hostAddition(int *A, int *B, int *C, int size) 
{ 
	for (int i = 0; i < size; i++) {
		for (int j = 0; j < size; j++) {
			C[i*size + j] = A[i*size + j] + B[i*size + j];
		}
	}
}//close hostaddition

__global__ void matrixAddition(int *A, int *B, int *C, int size) {
	int row = blockIdx.y*blockDim.y + threadIdx.y;
	int col = blockIdx.x*blockDim.x + threadIdx.x;

	if (row < size && col < size) {
		int temp = row * size + col;
		C[temp] = A[temp] + B[temp];
	}//close if
}
/**
 * @brief From "forvanya.txt"
 */

extern "C" void launch_matrix_multiply()
{

    /**
     * @brief Doing the matrix multiplication
     * 
     */
    time_t t;
    hipEvent_t start, stop, start1, stop1;

    hipEventCreate(&start);
	hipEventCreate(&start1);

    hipEventCreate(&stop);
	hipEventCreate(&stop1);

    float gpu_time = 0.0f, gpu_time1 = 0.0;

    size_t size = D*D*sizeof(int);

    //create pointers for host related stuff, allocate the memory required
	int *h_A = (int*)malloc(size);
	int *h_B = (int*)malloc(size);
	int *h_C = (int*)malloc(size);
	int *h_C1 = (int*)malloc(size);

	//create pointers for device related stuff, allocate the memory required
	int *d_A, *d_B, *d_C;
	hipMalloc((void**)&d_A, size);
	hipMalloc((void**)&d_B, size);
	hipMalloc((void**)&d_C, size);

	//seed that THICC BOI
	srand((unsigned)time(&t));

    //send in values into the host 2 input matrices
	for (int i = 0; i < D; i++) {
		for (int j = 0; j < D; j++) {
			int rand1 = rand() % 10;
			int rand2 = rand() % 10;
			*(h_A + i * D + j) = rand1;
			*(h_B + i * D + j) = rand2;
		}
	}

    //run it back baby, host addition start, stop and method call
	hipEventRecord(start, 0);
	hostAddition(h_A, h_B, h_C, D);
	hipEventRecord(stop, 0);

	//print out the results
	hipEventElapsedTime(&gpu_time, start, stop);
	printf("host addition time: %0.2f\n", gpu_time);

	//copy contents of host input matrices to the device
	hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    //setup threads per block and number of blocks.
	//should change D to just be strictly 16 later based on documentation ??...
	dim3 threadsPerBlock(TPB, TPB);
	dim3 numberOfBlocks(ceil(D / threadsPerBlock.x), ceil(D / threadsPerBlock.y));

	//addition by individual threads:
	hipEventRecord(start1, 0);
	matrixAddition <<<numberOfBlocks, threadsPerBlock>>>(d_A, d_B, d_C, D);
	hipEventRecord(stop1, 0);
	hipEventSynchronize(stop1);
	hipMemcpy(h_C1, d_C, size, hipMemcpyDeviceToHost);
	hipEventElapsedTime(&gpu_time1, start1, stop1);
	printf("\n normal matrix addition: %0.2f\n", gpu_time1);




}