#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "stdio.h"
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <sys/time.h>

#define N 16
#define USECPSEC 1000000ULL





__global__ void MultiBlock(int numThreads){
//find thread index so we can interate through 
int index = blockIdx.x * blockDim.x + threadIdx.x;

//seperate the threads into blocks
dim3 threadsPerBlock (1, 1);
dim3 numBlocks((N + threadsPerBlock.x -1) / threadsPerBlock.x, (N+threadsPerBlock.y -1) / threadsPerBlock.y);
//run through every block and every thread
for (int j = 0; numBlocks > j; j++){
    for (int i = 0; index > i ; i++){
        //vector at index, plus next vector
        vector_add<<<1, threadsPerBlock>>>();
    }
}
}

//TO-DO ADAM: needs to be made into global, cpu, etc.
//TIMING KERNEL EXECUTION WITH CPU TIMERS:
unsigned long long myCPUTimer(unsigned long long start=0){

  timeval tv;
  gettimeofday(&tv, 0);
  return ((tv.tv_sec*USECPSEC)+tv.tv_usec)-start;
} //returns time in microseconds 


int main(){
    //allocate host mem
    float *a, *b, *out;
    float *d_a, *d_b, *d_out; 

    //update size
    size_t size = D*D*sizeof(int);

 	int *h_A = (int*)malloc(size);
    int *h_C1 = (int*)malloc(size);

    int *d_A;
    hipMalloc((void**)&d_A, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

    MultiBlock<<<1, N>>>();

    //free mem
    hipFree(h_A);
    hipFree(h_C1);

    free(d_A); 
}