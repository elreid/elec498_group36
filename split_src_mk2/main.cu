/**
 * @file main.cu
 * @author Vanya Kootchin & Adam Bayley
 * @brief
 * @version 0.1
 * @date 2023-04-01
 *
 * @copyright Copyright (c) 2023
 *
 */
/**
 *
 *
 *
 */
/********* INITIALIZATION *********/
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <unistd.h>
#include <time.h>
#include <sys/time.h>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"


// #include "cuPrintf.cu"
/**
 *
 *
 *
 */
/***** GLOBAL VARIABLES *****/
#define TPB 16                // num threads in a block
#define D 1024                // num of elements in a row/column
#define N 16                  // num of elements in a matrix
#define USECPSEC 1000000ULL   // Idk what this is, cuda specific things
#define NUMPARTITIONS 10      // Number of partitions for OMPI?// TODO: Remove
#define NUMNODES 5            // Number of nodes in the graph
int CHECKSUM[NUMNODES] = {0}; // Global Checksum array
/**
 *
 *
 *
 */
/********* STRUCTURES *********/
struct node
{
    /**
     * Linked list node structure
     *
     */
    int buffer;
    int size;
    int partitions;
    struct node *next;
};

struct workload
{
    /**
     * GPU Workload Structure
     *
     */
    int *data_arr;
    int *check_sum;
    int id;
    int numnodes;
};
/**
 *
 *
 *
 */
/********* FUNCTIONS *********/
struct node *populate_list()
{
    /**
     *  Populate a linked list with the following values:
     *  - Returns a node structure that points to the head of the linked list
     */
    struct node *head = NULL;
    struct node *prev = NULL;

    for (int i = 0; i < NUMNODES; i += 1)
    {
        struct node *curr = (struct node *)malloc(sizeof(struct node));
        curr->buffer = 0x00000000 + i;
        curr->size = 256;
        curr->partitions = 16;
        curr->next = NULL;

        if (prev != NULL)
        {
            prev->next = curr;
        }
        else
        {
            head = curr;
        }

        prev = curr;
        if (i == NUMNODES - 1)
        {
            prev->next = NULL;
        }
    }
    return head;
}

void print_list(struct node *head)
{
    /**
     *  Print the linked list
     *
     */
    struct node *current = head;
    int i = 0;

    while (current != NULL)
    {
        printf("[%03d:%08X:%08X]:{buf:%08X,siz:%03d,par:%03d} ", i, current, current->next, current->buffer, current->size, current->partitions);
        if (current->next != NULL)
            printf(" -> \n");
        current = current->next;
        i++;
    }
    printf("\n");
}

void populate_array(struct node *head, int **arr)
{
    /**
     *  Make the array from the linekd list
     *  - Returns nothing, takes in a pointer to the head and a pointer to the array
     */
    struct node *current = head;

    free(*arr);
    *arr = (int *)malloc((NUMNODES * 3) * sizeof(int));
    if (*arr == NULL)
    {
        printf("ERR: Void array.\n");
        return;
    }
    else
    {
        for (int i = 0; i < NUMNODES * 3; i += 3)
        {
            (*arr)[i]       = current->buffer;
            (*arr)[i + 1]   = current->size;
            (*arr)[i + 2]   = current->partitions;
            current         = current->next;
        }
    }
}

void print_array(int *array, int length)
{
    /**
     * @ Print the array
     *
     */
    for (int i = 0; i < length; i++)
    {
        if (i % 3 == 0)
            printf("\n");
        printf("%08X ", array[i]);
    }
    printf("\n");
}

/**
 *
 *
 *
 */
/********* MAIN *********/
int main(int argc, char **argv)
{

    struct node *head = populate_list();
    print_list(head); // TODO: Remove this line

    // do host and device memory inits
    size_t size_list_arr = (NUMNODES * 3) * sizeof(int);

    int *h_list_arr = (int *)malloc(size_list_arr);

    populate_array(head, &h_list_arr);
    print_array(h_list_arr, NUMNODES * 3); // TODO: Remove this line

    return 0;
}