#include "hip/hip_runtime.h"
//Adam Bayley 20176309 19ahb Machine Problem 2
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>
#include <stdlib.h>
#include <ctime>

#define D 1024	//16x16, 256x256....
#define TPB 16	//num threads in a block

/*
__host__ void validateMatrix(int *m, int *h, int size) {
int success = 0;
for (int i = 0; i < size; i++) {
for (int j = 0; j < size; j++) {
if (m[i][j] != h[i][j]) {
success = 1;
}//close if
}//close for
}//close for
if (success == 0)
printf("Test Passed.\n");
else
printf("Test Failed.\n");
}//close validateMatrix
*/


void hostAddition(int *A, int *B, int *C, int size) { //might need __host__ ??....
	for (int i = 0; i < size; i++) {
		for (int j = 0; j < size; j++) {
			C[i*size + j] = A[i*size + j] + B[i*size + j];
		}//close j
	}//close i
}//close hostaddition

__global__ void matrixAddition(int *A, int *B, int *C, int size) {
	int row = blockIdx.y*blockDim.y + threadIdx.y;
	int col = blockIdx.x*blockDim.x + threadIdx.x;

	if (row < size && col < size) {
		int temp = row * size + col;
		C[temp] = A[temp] + B[temp];
	}//close if

}//close matrixAddition

__global__ void rowAddition(int *A, int *B, int*C, int size) {
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int y;
	if (i < size) {
		for (int k = 0; k < size; k++) {
			y = i*size + k;
			C[y] = B[y] + A[y];
		}//close for
	}//close if
}//close rowAddition

__global__ void colAddition(int *d_A, int *d_B, int*d_C, int size) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int y;
	if (i < size) {
		for (int k = 0; k < size; k++) {
			y = k*size + i;
			d_C[y] = d_B[y] + d_A[y];
		}//close for
	}//close if
}//close colAddition



int main() {

	//keep track of t for use with rand
	time_t t;

	//flags for each individual check (row, col, individual val)
	int correctFlag1 = 0;
	int correctFlag2 = 0;
	int correctFlag3 = 0;
	//int correctFlag[] = { 0,0,0 };

	//event variables
	hipEvent_t start, stop, start1, stop1, start2, stop2, start3, stop3;

	//create events for start times
	hipEventCreate(&start);
	hipEventCreate(&start1);
	hipEventCreate(&start2);
	hipEventCreate(&start3);

	//create events for stop times
	hipEventCreate(&stop);
	hipEventCreate(&stop1);
	hipEventCreate(&stop2);
	hipEventCreate(&stop3);
	hipDeviceSynchronize();

	//variables for difference between start and stop times
	float gpu_time = 0.0f, gpu_time1 = 0.0, gpu_time2 = 0.0f, gpu_time3 = 0.0f;

	//size of matrix calculation
	size_t size = D*D*sizeof(int);

	//create pointers for host related stuff, allocate the memory required
	int *h_A = (int*)malloc(size);
	int *h_B = (int*)malloc(size);
	int *h_C = (int*)malloc(size);
	int *h_C1 = (int*)malloc(size);
	int *h_C2 = (int*)malloc(size);
	int *h_C3 = (int*)malloc(size);


	//create pointers for device related stuff, allocate the memory required
	int *d_A, *d_B, *d_C;
	hipMalloc((void**)&d_A, size);
	hipMalloc((void**)&d_B, size);
	hipMalloc((void**)&d_C, size);

	//seed that THICC BOI
	srand((unsigned)time(&t));

	//send in values into the host 2 input matrices
	for (int i = 0; i < D; i++) {
		for (int j = 0; j < D; j++) {
			int rand1 = rand() % 10;
			int rand2 = rand() % 10;
			*(h_A + i * D + j) = rand1;
			*(h_B + i * D + j) = rand2;
		}
	}

	//run it back baby, host addition start, stop and method call
	hipEventRecord(start, 0);
	hostAddition(h_A, h_B, h_C, D);
	hipEventRecord(stop, 0);

	//print out the results
	hipEventElapsedTime(&gpu_time, start, stop);
	printf("host addition time: %0.2f\n", gpu_time);

	//copy contents of host input matrices to the device
	hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);


	//setup threads per block and number of blocks.
	//should change D to just be strictly 16 later based on documentation ??...
	dim3 threadsPerBlock(TPB, TPB);
	dim3 numberOfBlocks(ceil(D / threadsPerBlock.x), ceil(D / threadsPerBlock.y));

	//addition by individual threads:
	hipEventRecord(start1, 0);
	matrixAddition << < numberOfBlocks, threadsPerBlock >> >(d_A, d_B, d_C, D);
	hipEventRecord(stop1, 0);
	hipEventSynchronize(stop1);
	hipMemcpy(h_C1, d_C, size, hipMemcpyDeviceToHost);
	hipEventElapsedTime(&gpu_time1, start1, stop1);
	printf("\n normal matrix addition: %0.2f\n", gpu_time1);

	//addition by rows:
	hipEventRecord(start2, 0);
	rowAddition << <ceil(D / TPB), TPB >> >(d_A, d_B, d_C, D);
	hipEventRecord(stop2, 0);
	hipEventSynchronize(stop2);
	hipMemcpy(h_C2, d_C, size, hipMemcpyDeviceToHost);
	hipEventElapsedTime(&gpu_time2, start2, stop2);
	printf("\n Row time: %0.2f\n", gpu_time2);

	//addition by columns:
	hipEventRecord(start3, 0);
	colAddition << <ceil(D / TPB), TPB >> >(d_A, d_B, d_C, D);
	hipEventRecord(stop3, 0);
	hipEventSynchronize(stop3);
	hipMemcpy(h_C3, d_C, size, hipMemcpyDeviceToHost);
	hipEventElapsedTime(&gpu_time3, start3, stop3);
	printf("\n Column time: %0.2f\n", gpu_time3);

	//check if they stuff is equal
	for (int i = 0; i < D; i++) {
		for (int j = 0; j < D; j++) {
			if (*(h_C1 + i * D + j) != *(h_C + i * D + j))
				correctFlag1 = 1;
			if (*(h_C2 + i * D + j) != *(h_C + i * D + j))
				correctFlag2 = 1;
			if (*(h_C2 + i * D + j) != *(h_C + i * D + j))
				correctFlag3 = 1;
		}//end for j
	}//end for i

	if (correctFlag1 == 0)
		printf(" normal addition passed.\n");
	else
		printf(" normal addition failed.\n");

	if (correctFlag2 == 0)
		printf(" row addition passed.\n");
	else
		printf(" Row  addition failed.\n");

	if (correctFlag3 == 0)
		printf(" Column addition passed.\n\n");
	else
		printf(" Column addition failed.\n\n");

	//free host 
	hipHostFree(h_A);
	hipHostFree(h_B);
	hipHostFree(h_C);
	hipHostFree(h_C1);
	hipHostFree(h_C2);
	hipHostFree(h_C3);

	//free device
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

}
