#include "hip/hip_runtime.h"
//Adam Bayley 20176309 19ahb Question 1
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
//void PrintDeviceProperties: print off all the device properties for each device. 
void PrintDeviceProperties(hipDeviceProp_t dp) {
	printf("device name and type: %s \n", dp.name);
	printf(" clock rate: %d\n", dp.clockRate);
	printf("Total Global memory: %d\n ", dp.totalGlobalMem);
	printf("Total Constant memory: %d\n ", dp.totalConstMem);
	printf("Shared memory per block: %d\n ", dp.sharedMemPerBlock);
	printf("Warp size: %d\n ", dp.warpSize);
	printf("Number of registers available per block: %d\n", dp.regsPerBlock);
	printf("Max threads per block: %d\n ", dp.maxThreadsPerBlock);
	printf("Number of multiprocessors: %d\n", dp.multiProcessorCount);
	for (int i = 0; i < 3; ++i)
		printf("Maximum dimension %d of block:  %d\n", i, dp.maxThreadsDim[i]);
	for (int i = 0; i < 3; ++i)
		printf("Maximum dimension %d of grid:   %d\n", i, dp.maxGridSize[i]);
  //core calculation 
	int major = dp.major;
	int mpc = dp.multiProcessorCount;
	int cores = 0;
	switch (major) {
	case 2:
		cores = 32 * mpc;
		printf("This device has %d cores.\n", cores);
		break;
	case 3:
		cores = 192 * mpc;
		printf("This device has %d cores.\n", cores);
		break;
	case 5:
		cores = 128 * mpc;
		printf("This device has %d cores.\n", cores);
		break;
	case 6:
		cores = 64 * mpc;
		printf("This device has %d cores.\n", cores);
		break;
	case 7:
		cores = 64 * mpc;
		printf("This device has %d cores.\n", cores);
		break;
	case 8:
		cores = 64 * mpc;
		printf("This device has %d cores.\n", cores);
		break;
	default:
		cores = -1;
		printf("Error getting number of cores.\n");
		break;
	}//close case 
}//close void device info 
int main()
{
	int count;
	hipGetDeviceCount(&count);
	printf("there are %d devices\n", count);
	for (int i = 0; i < count; i++) {
		hipDeviceProp_t dp;
		hipGetDeviceProperties(&dp, i);
		printf("-------------------\n");
		PrintDeviceProperties(dp);
	}
}